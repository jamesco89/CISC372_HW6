#include "hip/hip_runtime.h"
/*
21S-CISC372-010
Homework 6 - Blurred cat
James Cooper

Simple optimized box blur
by: Greg Silber
Date: 5/1/2021
This program reads an image and performs a simple averaging of pixels within a supplied radius.  For optimization,
it does this by computing a running sum for each column within the radius, then averaging that sum.  Then the same for 
each row.  This should allow it to be easily parallelized by column then by row, since each call is independent.
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

/*
Part 3 Assignment:
More advanced CUDA
	Part 2 is kind of slow.
	To speed it up, allocate the memory we need on the device where possible
	Move that memory with hipMalloc and hipMemcpy up to device for calculation.
	Copy that memory back to the host in order to save it to the output file.
	Play with the values for blocksize to try to maximize performance.
-------------------------------------------------------------------------------------------   
Computes a single row of the destination image by summing radius pixels
Parameters: src: Teh src image as width*height*bpp 1d array
            dest: pre-allocated array of size width*height*bpp to receive summed row
            row: The current row number
	    height: The height of the source image 
            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
            rad: the width of the blur
            bpp: The bits per pixel in the src image
Returns: None
-------------------------------------------------------------------------------------------
*/
__global__
void computeRow(float* src, float* dest, int pWidth, int height, int radius, int bpp){
    int bradius = radius*bpp;
    int row = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if (height > row){
    	//initialize the first bpp elements so that nothing fails
    	for (int i = 0; i < bpp; i++)
        	dest[row*pWidth + i] = src[row*pWidth + i];
    
    	//start the sum up to radius*2 by only adding (nothing to subtract yet)
    	for (int i = bpp; i < bradius*2*bpp; i++)
        	dest[row*pWidth + i] = src[row*pWidth + i] + dest[row*pWidth + i - bpp];
    
    	for (int i = bradius*2 + bpp; i < pWidth; i++)
        	dest[row*pWidth + i] = src[row*pWidth + i] + dest[row*pWidth + i - bpp] - src[row*pWidth + i - 2*bradius - bpp];
   
    	//now shift everything over by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    	for (int i = bradius; i < pWidth; i++){
        	dest[row*pWidth + i - bradius] = dest[row*pWidth + i] / (radius*2+1);
    		}		

    	//now the first and last radius values make no sense, so blank them out
    	for (int i = 0; i < bradius; i++){
        	dest[row*pWidth + i] = 0;
        	dest[(row + 1)*pWidth - 1 - i] = 0;
    		}		
	}
} 
   
/*
--------------------------------------------------------------------------------------
Computes a single column of the destination image by summing radius pixels
Parameters: src: Teh src image as width*height*bpp 1d array
            dest: pre-allocated array of size width*height*bpp to receive summed row
            col: The current column number
            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
            height: The height of the source image
            radius: the width of the blur
            bpp: The bits per pixel in the src image
Returns: None
--------------------------------------------------------------------------------------
*/
__global__
void computeColumn(uint8_t* src, float* dest, int pWidth, int height, int radius, int bpp){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pWidth > col){
    	//initialize the first element of each column
    	dest[col] = src[col];
    
    	//start tue sum up to radius*2 by only adding
    	for (int i = 1; i <= radius*2; i++)
        	dest[i*pWidth + col] = src[i*pWidth + col] + dest[(i - 1)*pWidth + col];
    
    	for (int i = radius*2 + 1;i < height; i++)
        	dest[ i*pWidth + col] = src[i*pWidth + col] + dest[(i - 1)*pWidth + col] - src[(i - 2*radius - 1)*pWidth + col];
    
    	//now shift everything up by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    	for (int i = radius; i < height; i++){
        	dest[(i - radius)*pWidth + col] = dest[i*pWidth + col] / (radius*2 + 1);
    		}		

    	//now the first and last radius values make no sense, so blank them out
    	for (int i = 0; i < radius; i++){
        	dest[i*pWidth + col] = 0;
        	dest[(height - 1)*pWidth - i*pWidth + col] = 0;
    		}
	}	
}

/*
	Usage: Prints the usage for this program
	Parameters: name: The name of the program
	Returns: Always returns -1
*/

int Usage(char* name){
	printf("%s: <filename> <blur radius>\n\tblur radius=pixels to average on any side of the current pixel\n",name);
    	return -1;
}

int main(int argc,char** argv){
    float t1, t2;
    int radius = 0;
    int blockSize = 256;
    int numBlocks;
    int width, height, bpp, pWidth;
    char* filename;
    uint8_t *img, *destImg;
    float* hostDest, *dest, *mid;

    if (argc != 3)
        return Usage(argv[0]);
    filename = argv[1];
    sscanf(argv[2], "%d", &radius);

    // Start loading an image before processing 
    img = stbi_load(filename, &width, &height, &bpp, 0);   
    
    pWidth = width*bpp;  //actual width in bytes of an image row
    
    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMalloc(&mid, sizeof(float)*pWidth*height);
    hipMalloc(&dest,sizeof(float)*pWidth*height);
    hipMalloc(&destImg, sizeof(uint8_t)*pWidth*height);
    
    // Transfer data from host to device memory
    hipMemcpy(destImg, img, sizeof(uint8_t)*pWidth*height, hipMemcpyHostToDevice);
    
    stbi_image_free(img); //done with image

    // A clock() function to calculate the loading time of the image
    // Start counting 
    t1 = clock();
    
    numBlocks = (pWidth + blockSize - 1) / blockSize;
    
    // Excecuting a computeColumn kernel
    computeColumn<<<numBlocks, blockSize>>>(destImg, mid, pWidth, height, radius, bpp);
    
    //Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    // Allocate Unified Memory -- accessible from CPU or GPU
    //hipMallocManaged(&img, sizeof(uint8_t)*pWidth*height);    

    numBlocks = (height + blockSize - 1) / blockSize;
    
    // Excecuting a computeRow kernel
    computeRow<<<numBlocks, blockSize>>>(mid, dest, pWidth, height, radius, bpp);
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();   
    
    hipFree(mid);
    
    // End counting
    t2 = clock();
  
    hostDest = (float*)malloc(sizeof(float)*pWidth*height);
    hipMemcpy(hostDest, dest, sizeof(float)*pWidth*height, hipMemcpyDeviceToHost);
    
    hipFree(dest);	 
    
    // Now back to int8 so we can save it
    img = (uint8_t*)malloc(sizeof(uint8_t)*pWidth*height);
    for (int i = 0; i < pWidth*height; i++){
        img[i] = (uint8_t)hostDest[i];
    	}
    
    free(hostDest);
    
    // Display the result of the image after applying a gauss blur method
    stbi_write_png("output.png", width, height, bpp, img, bpp*width);
    
     free(img);
    
    // Show the time to complete the image after processing with the radius we desired
    printf("Blur with radius %d complete in %f seconds\n", radius, (t2 - t1) / CLOCKS_PER_SEC);
    
    hipFree(destImg);

}
