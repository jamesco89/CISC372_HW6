#include "hip/hip_runtime.h"
//Simple optimized box blur
//by: Greg Silber
//Date: 5/1/2021
//This program reads an image and performs a simple averaging of pixels within a supplied radius.  For optimization,
//it does this by computing a running sum for each column within the radius, then averaging that sum.  Then the same for 
//each row.  This should allow it to be easily parallelized by column then by row, since each call is independent.

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

/*   
Computes a single row of the destination image by summing radius pixels
Parameters: src: Teh src image as width*height*bpp 1d array
            dest: pre-allocated array of size width*height*bpp to receive summed row
            row: The current row number
            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
            rad: the width of the blur
            bpp: The bits per pixel in the src image
Returns: None

           Each row runs in its own thread
           Thread block size = 256
           computeRow into a kernel, and figuring out the col parameter from the
           - threadIdx
           - blockIdx
           - blockDim
           Sync up the threads with a call to cudaDeviceSync
           Convert back to unit8_t array, and save the image
           Use cuda MallocManaged(...) and hipFree(...) for all arrays
           A block size of 256 -> a block acount of(width + 255)/256 coloumns
           Check in kernel funtion for unsued threads where the computed coloum > pWidth.
           Do the same for the rows (height + 255)/256
*/
__global__
void computeRow(float* src, float* dest, int row, int pWidth, int radius, int bpp){
    /*
           Each row  runs in its own thread
           Thread block size = 256
           computeColumn into a kernel, and figuring out the col parameter from the
           - threadIdx
           - blockIdx
           - blockDim
           Sync up the threads with a call to cudaDeviceSync and repeat the process for each row
           Convert back to unit8_t array, and save the image
           Use cuda MallocManaged(...) and hipFree(...) for all arrays
           A block size of 256 -> a block acount of(width + 255)/256 coloumns
           Check in kernel funtion for unsued threads where the computed coloum > pWidth.
           Do the same for the rows (height + 255)/256
    */

    int bradius = radius*bpp;
    int row = blockIdx.x * blockDim.x + threadIndex.x; 
    
    if (height > row){
    //initialize the first bpp elements so that nothing fails
    for (int i = 0; i < bpp; i++)
        dest[row*pWidth + i] = src[row*pWidth + i];
    
    //start the sum up to radius*2 by only adding (nothing to subtract yet)
    for (int i = bpp; i < bradius*2*bpp; i++)
        dest[row*pWidth + i] = src[row*pWidth + i] + dest[row*pWidth + i - bpp];
    
    for (int i = bradius*2 + bpp; i < pWidth; i++)
        dest[row*pWidth + i] = src[row*pWidth + i] + dest[row*pWidth + i - bpp] - src[row*pWidth + i - 2*bradius - bpp];
   
    //now shift everything over by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    for (int i = bradius; i < pWidth; i++){
        dest[row*pWidth + i - bradius] = dest[row*pWidth + i] / (radius*2+1);
    }

    //now the first and last radius values make no sense, so blank them out
    for (int i = 0; i < bradius; i++){
        dest[row*pWidth + i] = 0;
        dest[(row + 1)*pWidth - 1 - i] = 0;
    }
}

//Computes a single column of the destination image by summing radius pixels
//Parameters: src: Teh src image as width*height*bpp 1d array
//            dest: pre-allocated array of size width*height*bpp to receive summed row
//            col: The current column number
//            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
//            height: The height of the source image
//            radius: the width of the blur
//            bpp: The bits per pixel in the src image
//Returns: None
__global__
void computeColumn(uint8_t* src, float* dest, int col, int pWidth, int height, int radius, int bpp){
	/*
	   Each column runs in its own thread
	   Thread block size = 256
	   computeColumn into a kernel, and figuring out the col parameter from the
	   - threadIdx
	   - blockIdx
	   - blockDim
	   Sync up the threads with a call to cudaDeviceSync and repeat the process for each row
	   Convert back to unit8_t array, and save the image
	   Use cuda MallocManaged(...) and hipFree(...) for all arrays
	   A block size of 256 -> a block acount of(width + 255)/256 coloumns
	   Check in kernel funtion for unsued threads where the computed coloum > pWidth.
	   Do the same for the rows (height + 255)/256
	 */
    int i;
    
    //initialize the first element of each column
    dest[col] = src[col];
    
    //start tue sum up to radius*2 by only adding
    for (i = 1; i <= radius*2; i++)
        dest[i*pWidth + col] = src[i*pWidth + col] + dest[(i - 1)*pWidth + col];
    
    for (i = radius*2 + 1;i < height; i++)
        dest[ i*pWidth + col] = src[i*pWidth + col] + dest[(i - 1)*pWidth + col] - src[(i - 2*radius - 1)*pWidth + col];
    
    //now shift everything up by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    for (i = radius; i < height; i++){
        dest[(i - radius)*pWidth + col] = dest[i*pWidth + col] / (radius*2 + 1);
    }

    //now the first and last radius values make no sense, so blank them out
    for (i = 0; i < radius; i++){
        dest[i*pWidth + col] = 0;
        dest[(height - 1)*pWidth - i*pWidth + col] = 0;
    }
}

//Usage: Prints the usage for this program
//Parameters: name: The name of the program
//Returns: Always returns -1
int Usage(char* name){
    printf("%s: <filename> <blur radius>\n\tblur radius=pixels to average on any side of the current pixel\n",name);
    return -1;
}

int main(int argc,char** argv){
    float t1,t2;
    int radius=0;
    int i;
    int width, height, bpp, pWidth;
    char* filename;
    uint8_t *img;
    float* dest, *mid;

    if (argc != 3)
        return Usage(argv[0]);
    filename = argv[1];
    sscanf(argv[2], "%d", &radius);
   
    img = stbi_load(filename, &width, &height, &bpp, 0);

    //mid = malloc(sizeof(float)*pWidth*height);   
    //dest = malloc(sizeof(float)*pWidth*height);   
    
    pWidth = width*bpp;  //actual width in bytes of an image row
    hipMallocManaged(&mid, sizeof(float)*pWidth*height);
    hipMallocManaged(&dest, sizeof(float)*pWidth*height;

    stbi_image_free(img); //done with image

    t1 = clock();
    int blockSize = 256;
    gridSize = (pWidth + blockSize - 1) / blockSize;
    computeColumn<<<gridSize, blockSize>>>(dest, mid, pWidth, height, radius, bpp);
    //Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    blockSize = 256;
    gridSize = (height + blockSize - 1) / blockSize;
    computeRow(mid, dest pWidth,heigh, radius, bpp);
    //Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    hipFree(mid);
    t2 = clock();

    //now back to int8 so we can save it
    img = malloc(sizeof(uint8_t)*pWidth*height);
    for (i = 0;i < pWidth*height; i++){
        img[i]=(uint8_t)dest[i];
    }

    free(dest);   
    stbi_write_png("output.png", width, height, bpp, img, bpp*width);
    free(img);
    printf("Blur with radius %d complete in %ld seconds\n", radius, (t2 - t1)/CLOCKS_PER_SE);
}
